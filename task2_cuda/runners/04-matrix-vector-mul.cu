#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <MatrixVectorMul.cuh>
#include <vector>

int main() {
    int height = 1000;
    int width = 1000;

    float* A   = (float*) malloc(height * width * sizeof(float));
    float* vec = (float*) malloc(width * sizeof(float));
    float* res = (float*) malloc(height * sizeof(float));

    for (int i = 0; i < height; ++i)
        for (int j = 0; j < width; ++j)
            A[i * width + j] = 1.0;

    for (int i = 0; i < width; ++i) {
        vec[i] = 2;
        res[i] = 0;
    }

    float* A_dev = nullptr;
    float* vec_dev = nullptr;
    float* res_dev = nullptr;

    hipMalloc(&A_dev, height * width * sizeof(float));
    hipMalloc(&vec_dev, width * sizeof(float));
    hipMalloc(&res_dev, height * sizeof(float));

    hipMemcpy(A_dev, A, height * width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vec_dev, vec, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(res_dev, res, width * sizeof(float), hipMemcpyHostToDevice);

    std::vector <float> times;

    for (int n = 1; n < 1024; ++n) {
        int height = n;
        int width = n;

        float* A_dev = nullptr;
        float* vec_dev = nullptr;
        float* res_dev = nullptr;

        hipMalloc(&A_dev, height * width * sizeof(float));
        hipMalloc(&vec_dev, width * sizeof(float));
        hipMalloc(&res_dev, height * sizeof(float));

        float time = 0;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        int threadsPerBlock = 256;
        int blocksPerGrid = (width + threadsPerBlock - 1) / threadsPerBlock;

        MatrixVectorMul <<<threadsPerBlock, blocksPerGrid, threadsPerBlock * sizeof(float)>>>(height, width, A_dev,
                                                                                              vec_dev, res_dev);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        times.push_back(time);
    }

    for (auto& time : times) {
        std::cout << time << " ";
    }

    hipMemcpy(res, res_dev, width * sizeof(float), hipMemcpyDeviceToHost);

    int error = 0;

    for (int i = 0; i < height; ++i) {
        if (fabs(res[i] - 2*width) > 0.1)
            ++error;
    }

    free(A);
    free(vec);
    free(res);

    hipFree(A_dev);
    hipFree(vec_dev);
    hipFree(res_dev);

    std::cout << error << " ";
    std::cout << time;

    return 0;
}

