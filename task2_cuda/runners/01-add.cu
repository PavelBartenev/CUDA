#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include "KernelAdd.cuh"
#include <vector>


void measure_time_threads() {
    int N = 1 << 20;
    int size = N * sizeof(float);

    float* device_A = nullptr;
    float* device_B = nullptr;
    float* device_C = nullptr;
    hipMalloc(&device_A, size);
    hipMalloc(&device_B, size);
    hipMalloc(&device_C, size);

    std::vector <float> times;

    for (int threadsPerBlock = 1; threadsPerBlock < 512; ++threadsPerBlock) {
        float time = 0;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        KernelAdd <<<blocksPerGrid, threadsPerBlock>>>(N, device_A, device_B, device_C);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        times.push_back(time);
    }

    for (auto& time : times) {
        std::cout << time << " ";
    }
}

void measure_time_vec_size() {
    int N = 1 << 20;
    int size = N * sizeof(float);

    float* device_A = nullptr;
    float* device_B = nullptr;
    float* device_C = nullptr;
    hipMalloc(&device_A, size);
    hipMalloc(&device_B, size);
    hipMalloc(&device_C, size);

    std::vector <float> times;

    int threadsPerBlock = 256;

    for (int n = 1; n < N; n += 100) {
        float time = 0;
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        KernelAdd <<<blocksPerGrid, threadsPerBlock>>>(n, device_A, device_B, device_C);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        times.push_back(time);
    }

    for (auto& time : times) {
        std::cout << time << " ";
    }
}


int main() {
    int64_t N = 1 << 20;
    size_t size = N * sizeof(float);

    float* host_A = (float*)malloc(size);
    float* host_B = (float*)malloc(size);
    float* host_C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        host_A[i] = 1;
        host_B[i] = 2;
    }

    float* device_A = nullptr;
    float* device_B = nullptr;
    float* device_C = nullptr;
    hipMalloc(&device_A, size);
    hipMalloc(&device_B, size);
    hipMalloc(&device_C, size);

    hipMemcpy(device_A, host_A, size, hipMemcpyHostToDevice);
    hipMemcpy(device_B, host_B, size, hipMemcpyHostToDevice);
    hipMemcpy(host_C, device_C, size, hipMemcpyDeviceToHost);

    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    int error = 0;
    std::cout << N << "\n";

    for (int i = 0; i < N; ++i) {
        if (fabs(host_C[i] - 3) > 0.01){
            error+=1;
        }
    }

    std::cout << error;

    free(host_A);
    free(host_B);
    free(host_C);

    return 0;
}


